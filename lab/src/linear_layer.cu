#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <assert.h>

#include "linear_layer.h"
#include "nn_exception.h"

using namespace std;
using namespace cv;

__global__ void linearLayerForward(float *W, float* input, float* output, float* b,
									const int W_rows, const int W_cols,
									const int input_rows, const int input_cols) 
{
    //TODO: complete the linear layer forward propagation
}

__global__ void linearLayerBackprop(float *W, float* eB, float* eA,
									const int W_rows, const int W_cols,
									const int eB_rows, const int eB_cols) 
{
    //TODO: complete the linear layer backpropagation
}

__global__ void linearLayerUpdateWeights(float *eB, float* input, float* W,
									const int eB_rows, const int eB_cols,
									const int input_rows, const int input_cols, float learning_rate)
{
    //TODO: complete the gradient descent for weight updates
}

__global__ void linearLayerUpdateBias(float *eB, float* b,
									const int eB_rows, const int eB_cols,
									const int b_rows, float learning_rate)
{
    //TODO: complete the gradient descent for bias updates
}

LinearLayer::LinearLayer(string name, Shape W_shape) 
{
	W_shape.transpose();
	
	Matrix weights(W_shape);
	Matrix bias(W_shape.rows, 1);

	this->W = weights;
	this->b = bias;

	this->name = name;
	b.allocateMemory();
	W.allocateMemory();
	initializeBiasWithZeros();
	initializeWeightsRandomly();
}

LinearLayer::~LinearLayer() {}

void LinearLayer::initializeWeightsRandomly() 
{
	
	float mean = 0.0;
	float stddev = 1.0;

	theRNG().state = time(NULL);
	randn(W.data_host, Scalar(mean), Scalar(stddev));

	W.copyHostToDevice();
}

void LinearLayer::initializeWeightsHalf() 
{
	W.data_host = Scalar(0.5f);

	W.copyHostToDevice();
}


void LinearLayer::initializeBiasWithZeros()
{
	
	b.data_host = Scalar(0.0f);

	b.copyHostToDevice();
}

Matrix& LinearLayer::forward(Matrix& input)
{
//	printf("W shape : (%lu %lu)\n", W.shape.rows, W.shape.cols);
//	printf("input shape : (%lu %lu)\n", input.shape.rows, input.shape.cols);

	assert(W.shape.cols == input.shape.rows);
	
	this->input = input;

	Shape output_shape(W.shape.rows, input.shape.cols);

	output.allocateMemoryIfNotAllocated(output_shape);

	computeAndStoreLayerOutput(input);
	NNException::throwIfDeviceErrorsOccurred("Cannot perform linear layer forward propagation");

	return output;
}

void LinearLayer::computeAndStoreLayerOutput(Matrix& input) {
	dim3 block_size(8, 8);
	dim3 num_of_blocks(	(output.shape.cols + block_size.x - 1) / block_size.x,
						(output.shape.rows + block_size.y - 1) / block_size.y);

	linearLayerForward<<<num_of_blocks, block_size>>>( W.data_device,
															input.data_device,
															output.data_device,
															b.data_device,
															W.shape.rows, W.shape.cols,
															input.shape.rows, input.shape.cols);
}

Matrix& LinearLayer::backprop(Matrix& eB, float learning_rate)
{
	eA.allocateMemoryIfNotAllocated(input.shape);

	computeAndStoreBackpropError(eB);
	NNException::throwIfDeviceErrorsOccurred("Cannot perform back propagation.");

	updateBias(eB, learning_rate);
	NNException::throwIfDeviceErrorsOccurred("Cannot perform bias update.");

	updateWeights(eB, learning_rate);
	NNException::throwIfDeviceErrorsOccurred("Cannot perform weights update.");

	return eA;
}


void LinearLayer::computeAndStoreBackpropError(Matrix& eB) {
	dim3 block_size(8, 8);
	dim3 num_of_blocks(	(input.shape.cols + block_size.x - 1) / block_size.x,
						(input.shape.rows + block_size.y - 1) / block_size.y);

	linearLayerBackprop<<<num_of_blocks, block_size>>>( W.data_device,
															eB.data_device,
															eA.data_device,
															W.shape.rows, W.shape.cols,
															eB.shape.rows, eB.shape.cols);
}

void LinearLayer::updateWeights(Matrix& eB, float learning_rate) {
	dim3 block_size(8, 8);
	dim3 num_of_blocks(	(W.shape.cols + block_size.x - 1) / block_size.x,
						(W.shape.rows + block_size.y - 1) / block_size.y);

	linearLayerUpdateWeights<<<num_of_blocks, block_size>>>(eB.data_device,
															input.data_device,
															W.data_device,
															eB.shape.rows, eB.shape.cols,
															input.shape.rows, input.shape.cols,
															learning_rate);
}

void LinearLayer::updateBias(Matrix& eB, float learning_rate) {
	dim3 block_size(256);
	dim3 num_of_blocks( (eB.shape.rows * eB.shape.cols + block_size.x - 1) / block_size.x);

	linearLayerUpdateBias<<<num_of_blocks, block_size>>>(eB.data_device,
															b.data_device,
															eB.shape.rows, eB.shape.cols,
															b.shape.rows, learning_rate);
}

int LinearLayer::getXDim() const {
	return W.shape.cols;
}

int LinearLayer::getYDim() const {
	return W.shape.rows;
}

Matrix LinearLayer::getWeightsMatrix() const {
	return W;
}

Matrix LinearLayer::getBiasVector() const {
	return b;
}
